#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <cstdint>
#define BLOCK_SIZE 16
typedef struct
{
int width;
int height;
float* dataArray;
int dataArraySize;
} Data;
__global__ void VecAdd(Data dim_gpu, float* values_gpu) {
const int index = threadIdx.x + blockIdx.x * blockDim.x;
if (index < dim_gpu.dataArraySize) {
values_gpu[index] = dim_gpu.dataArray[index] + 100.0;
}
}
int main()
{
constexpr int rgb_size = 3;
Data dim{};
dim.width = 400* rgb_size;
dim.height = 400;
dim.dataArraySize = dim.width * dim.height * rgb_size;
dim.dataArray = new float[dim.dataArraySize];
float mem_size = dim.width * dim.height * rgb_size * sizeof(float);
for (int i = 0; i < dim.dataArraySize; i++)
{
dim.dataArray[i] = float((rand() % 100) * 1.59); // random as: 0-99 * 1.59
if (i % 48000 == 0) {
printf("%.3f\n", dim.dataArray[i]);
}
}
// tutaj chcemy zwracać wartości, więc potrzeba nam pamięci po cpu i gpu
float* values_cpu = new float[dim.dataArraySize];
float* values_gpu;
hipMalloc(&values_gpu, mem_size);
Data dim_gpu{};
dim_gpu.width = dim.width;
dim_gpu.height = dim.height;
dim_gpu.dataArraySize = dim.dataArraySize;
hipMalloc(&dim_gpu.dataArray, mem_size);
hipMemcpy(dim_gpu.dataArray, dim.dataArray, mem_size, hipMemcpyHostToDevice);
int threadsPerBlock = 256;
int blocksPerGrid =
(dim.dataArraySize + threadsPerBlock - 1) / threadsPerBlock;
VecAdd <<<blocksPerGrid, threadsPerBlock >>> (dim_gpu, values_gpu);
hipMemcpy(values_cpu, values_gpu, mem_size, hipMemcpyDeviceToHost);
for (int i = 0; i < dim.dataArraySize; i++)
{
if (i % 48000 == 0 ) {
printf("%.3f %.3f\n", (dim.dataArray[i] + 100.0), values_cpu[i]);
}
}
hipError_t error = hipGetLastError();
if (error != hipSuccess)
{
fprintf(stderr, "ERROR: %s\n", hipGetErrorString(error));
exit(-1);
}
//zwracamy pamięć
delete(dim.dataArray);
delete(values_cpu);
hipFree(dim_gpu.dataArray);
hipFree(values_gpu);
return 0;
}S
